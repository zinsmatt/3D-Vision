#include "hip/hip_runtime.h"
#include "kernels.cuh"

#include <stdio.h>
#include <cmath>
#include <iostream>
#include <fstream>

#include "constants.h"


__device__
inline double getBilinearInterpolatedValue_cuda(const unsigned char *img, double pt[2]) {
    const unsigned char* d = &img[(int)pt[1] * width + (int)pt[0]];
    double xx = pt[0] - floor(pt[0]);
    double yy = pt[1] - floor(pt[1]);
    return ((1 - xx) * (1 - yy) * double(d[0]) +
            xx * (1 - yy) * double(d[1]) +
            (1 - xx) * yy * double(d[width]) +
            xx * yy * double(d[width + 1])) / 255.0;
}

__device__
inline void pix2cam_cuda(const double in[2], double out[3]) {
    out[0] = (in[0] - cx) / fx;
    out[1] =  (in[1] - cy) / fy;
    out[2] = 1.0;
}

__device__
inline void cam2pix_cuda(const double in[3], double out[2]) {
    out[0] = in[0] * fx / in[2] + cx;
    out[1] = in[1] * fy / in[2] + cy;
}

__device__
double norm3_cuda(const double in[3])
{
    return sqrt(in[0]*in[0] + in[1]*in[1] + in[2]*in[2]);
}

__device__
double norm2_cuda(const double in[2])
{
    return sqrt(in[0]*in[0] + in[1]*in[1]);
}


// inplace normalization vec 3
__device__
inline void normalize3_cuda(double in_out[3]) {
    double d = sqrt(in_out[0]*in_out[0] 
                    + in_out[1]*in_out[1]
                    + in_out[2]*in_out[2]);
    in_out[0] /= d;
    in_out[1] /= d;
    in_out[2] /= d;
}

// inplace normalization vec 2
__device__
inline void normalize2_cuda(double in_out[2]) {
    double d = sqrt(in_out[0]*in_out[0] + in_out[1]*in_out[1]);
    in_out[0] /= d;
    in_out[1] /= d;
}

__device__
void transform_cuda(double x[3], const double T[12], double out[3])
{
    for (int i = 0; i < 3; ++i)
    {
        out[i] = x[0] * T[i*4] + x[1] * T[i*4+1] + x[2] * T[i*4+2] +  T[i*4+3];
    }
}


__device__
double ZNCC_cuda(const unsigned char *im1, const double pt1[2], const unsigned char *im2, const double pt2[2])
{
    // no need to consider block partly outside because of boarder
    double v1[ncc_area], v2[ncc_area];
    double s1 = 0.0, s2 = 0.0;
    int idx = 0;
    for (int i = -ncc_window_size; i <= ncc_window_size; ++i)
    {
        for (int j = -ncc_window_size; j <= ncc_window_size; ++j)
        {
            double val_1 = ((double) im1[((int)pt1[1] + i) * width + (int)pt1[0] + j]) / 255;
            double temp_p2[2] = {pt2[0] + j, pt2[1] + i};
            double val_2 = getBilinearInterpolatedValue_cuda(im2, temp_p2);
            s1 += val_1;
            s2 += val_2;
            v1[idx] = val_1;
            v2[idx] = val_2;
            ++idx;
        }
    }

    double mean_1 = s1 / ncc_area;
    double mean_2 = s2 / ncc_area;

    double numerator = 0.0;
    double den1 = 0.0, den2 = 0.0;
    for (int i = 0; i < ncc_area; ++i)
    {
        double zv1 = v1[i] - mean_1;
        double zv2 = v2[i] - mean_2;
        numerator += zv1*zv2;
        den1 += zv1 * zv1;
        den2 += zv2 * zv2;
    }
    auto zncc =  numerator / (sqrt(den1 * den2 + epsilon));
    // std::cout << "zncc = " << zncc << "\n";
    return zncc;
}

__device__
bool epipolar_search_cuda(const unsigned char* ref, const unsigned char* cur, 
                          const double Tcr[12], const double pt[2],
                          double depth_mu, double depth_sigma2, 
                          double best_pc[2], double epipolar_dir[2])
{

    double depth_sigma = sqrt(depth_sigma2);
    double dmax = depth_mu + 3 * depth_sigma;
    double dmin = depth_mu - 3 * depth_sigma;
    dmin = max(0.1, dmin);

    double pn[3];
    pix2cam_cuda(pt, pn);
    normalize3_cuda(pn);
    double P_max[3] = {pn[0] * dmax, pn[1] * dmax, pn[2] * dmax};
    double P_min[3] = {pn[0] * dmin, pn[1] * dmin, pn[2] * dmin};
    double P_mu[3] = {pn[0] * depth_mu, pn[1] * depth_mu, pn[2] * depth_mu};

    double P_max_cur[3], P_min_cur[3], P_mu_cur[3];
    transform_cuda(P_max, Tcr, P_max_cur);
    transform_cuda(P_min, Tcr, P_min_cur);
    transform_cuda(P_mu, Tcr, P_mu_cur);


    double pc_max[2], pc_min[2], pc_mu[2];
    cam2pix_cuda(P_max_cur, pc_max);
    cam2pix_cuda(P_min_cur, pc_min);
    cam2pix_cuda(P_mu_cur, pc_mu);


    double epipolar_line[2] = {pc_max[0] - pc_min[0], pc_max[1] - pc_min[1]};
    epipolar_dir[0] = epipolar_line[0];
    epipolar_dir[1] = epipolar_line[1];
    normalize2_cuda(epipolar_dir);
    double epipolar_line_norm = norm2_cuda(epipolar_line);

    // double step = 0.7;
    // int nb_samples = std::ceil(epipolar_line.norm() / step);

    double half_range = 0.5 * epipolar_line_norm;
    if (half_range > 100) half_range = 100;

    double best_zncc = -1.0;
    for (double l = -half_range; l<= half_range; l+= 0.7)
    {
        double p[2] = {pc_mu[0] + l * epipolar_dir[0], pc_mu[1] + l * epipolar_dir[1]};

        if (p[0] < boarder || p[0] >= width-boarder || p[1] < boarder || p[1] >= height-boarder)
            continue; // p is outside the cur image

        double zncc = ZNCC_cuda(ref, pt, cur, p);
        if (zncc > best_zncc)
        {
            best_zncc = zncc;
            best_pc[0] = p[0];
            best_pc[1] = p[1];
        }
    }
    if (best_zncc < 0.85)
        return false;
    else
        return true;
}

__device__
double dot3_cuda(const double a[3], const double b[3])
{
    return a[0]*b[0] + a[1]*b[1] + a[2]*b[2];
}

__device__
double det2_cuda(const double A[2][2])
{
    return A[0][0] * A[1][1] - A[1][0] * A[0][1];
}

__device__
void solve_Axb2_cuda(const double A[2][2], const double b[2], double res[2])
{
    double det_inv = 1.0 / det2_cuda(A);
    double A_inv[2][2];
    A_inv[0][0] = det_inv * A[1][1];
    A_inv[0][1] = -det_inv * A[0][1];
    A_inv[1][0] = -det_inv * A[1][0];
    A_inv[1][1] = det_inv * A[0][0];

    res[0] = A_inv[0][0] * b[0] + A_inv[0][1] * b[1];
    res[1] = A_inv[1][0] * b[0] + A_inv[1][1] * b[1];
}

__device__
void update_depth_filter_cuda(const double pr[2], const double pc[2], const double Trc[12], const double epipolar_dir[2], double *depth, double *cov2)
{
    double fr[3];
    pix2cam_cuda(pr, fr);
    normalize3_cuda(fr);

    double fc[3];
    pix2cam_cuda(pc, fc);
    normalize3_cuda(fc);
    
    double f2[3] = {dot3_cuda(Trc, fc),
                    dot3_cuda(Trc+4, fc),
                    dot3_cuda(Trc+8, fc)};

    double trc[3] = {Trc[3], Trc[7], Trc[11]};
    double A[2][2];
    double b[2];

    A[0][0] = dot3_cuda(fr, fr);
    A[0][1] = dot3_cuda(fr, f2);
    A[1][0] = dot3_cuda(f2, fr);
    A[1][1] = dot3_cuda(f2, f2);
    A[0][1] *= -1;
    A[1][1] *= -1;
    
    b[0] = dot3_cuda(fr, trc);
    b[1] = dot3_cuda(f2, trc);

    if (abs(det2_cuda(A)) < 1e-20) // not invertible
        return;

    double res[2];
    solve_Axb2_cuda(A, b, res);
    double P1[3] = {fr[0] * res[0], fr[1] * res[0], fr[2] * res[0]};
    double P2[3] = {trc[0] + fc[0] * res[1], trc[1] + fc[1] * res[1], trc[2] + fc[2] * res[1]};
    double P_est[3] = {(P1[0] + P2[0]) * 0.5, 
                       (P1[1] + P2[1]) * 0.5, 
                       (P1[2] + P2[2]) * 0.5};
    double depth_obs = norm3_cuda(P_est);

    double P[3] = {fr[0] * depth_obs, fr[1] * depth_obs, fr[2] * depth_obs};
    double a[3] = {P[0] - trc[0], P[1] - trc[1], P[2] - trc[2]};

    double t[3] = {trc[0], trc[1], trc[2]};
    normalize3_cuda(t);

    double alpha = acos(dot3_cuda(fr, t));
    double beta = acos(-dot3_cuda(a, t) / norm3_cuda(a));

    double pc2[2] = {pc[0] + epipolar_dir[0], pc[1] + epipolar_dir[1]};
    double fc2[3];
    pix2cam_cuda(pc2, fc2);
    normalize3_cuda(fc2);
    double beta_2 = acos(-dot3_cuda(fc2, t));

    double gamma = M_PI - alpha - beta_2;
    double d_noise = norm3_cuda(trc) * sin(beta_2) / sin(gamma); // sinus law
    double sigma_obs = depth_obs - d_noise;
    double sigma2_obs = sigma_obs * sigma_obs;


    // Depth fusion
    double d = depth[(int)pr[1] * width + (int)pr[0]];
    double sigma2 = cov2[(int)pr[1] * width + (int)pr[0]];

    double d_fused = (sigma2_obs * d + sigma2 * depth_obs) / (sigma2 + sigma2_obs);
    double sigma2_fused = (sigma2 * sigma2_obs) / (sigma2 + sigma2_obs);

    depth[(int)pr[1] * width + (int)pr[0]] = d_fused;
    cov2[(int)pr[1] * width + (int)pr[0]] = sigma2_fused;

}


__device__ double Tcr_global[12];
__device__ double Trc_global[12];


__global__
void process_pixel_cuda(const unsigned char* ref, const unsigned char* cur, double *depth, double *cov2)
{

    int j = boarder + (blockIdx.x * blockDim.x) + threadIdx.x;
    int i = boarder + (blockIdx.y * blockDim.y) + threadIdx.y;

    double depth_mu = depth[i*width+j];
    double depth_sigma2 = cov2[i*width+j];

    if (depth_sigma2 < min_cov || depth_sigma2 > max_cov)
        return;

    double pr[2] = {(double)j, (double)i};
    double pc[2];
    double epipolar_dir[2];

    bool found = epipolar_search_cuda(ref, cur, Tcr_global, pr, depth_mu, depth_sigma2, pc, epipolar_dir);
    
    if (!found)
        return;

    update_depth_filter_cuda(pr, pc, Trc_global, epipolar_dir, depth, cov2);
}


void wrapper_update_cuda(const unsigned char* ref, const unsigned char* cur, double Tcr[3][4], double Trc[3][4], double *depth, double *cov2)
{

    size_t size_uchar = sizeof(unsigned char) * width * height;
    size_t size_double = sizeof(double) * width * height;
    unsigned char *ref_cuda, *cur_cuda;
    hipMalloc(&ref_cuda, size_uchar);
    hipMalloc(&cur_cuda, size_uchar);

    double *depth_cuda, *cov2_cuda;
    hipMalloc(&depth_cuda, size_double);
    hipMalloc(&cov2_cuda,  size_double);

    hipMemcpy(ref_cuda, ref, size_uchar, hipMemcpyHostToDevice);
    hipMemcpy(cur_cuda, cur, size_uchar, hipMemcpyHostToDevice);
    
    hipMemcpy(depth_cuda, depth, size_double, hipMemcpyHostToDevice);
    hipMemcpy(cov2_cuda, cov2, size_double, hipMemcpyHostToDevice);


    int A = 480 - 2 * boarder; // height
    int B = 640 - 2 * boarder; // width

    dim3 block_dim(16, 16);
    dim3 grid_dim(B / 16 + 1, A / 16 + 1);
    // std::cout << "grid_dim " << grid_dim.x << " " << grid_dim.y << "\n";
    // std::cout << "block_dim " << block_dim.x << " " << block_dim.y << "\n";
   

    hipMemcpyToSymbol(HIP_SYMBOL(Tcr_global), &Tcr[0][0], 12 * sizeof(double), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(Trc_global), &Trc[0][0], 12 * sizeof(double), 0, hipMemcpyHostToDevice);

    process_pixel_cuda<<<grid_dim, block_dim>>>(ref_cuda, cur_cuda, depth_cuda, cov2_cuda);
    hipDeviceSynchronize();


    hipMemcpy(depth, depth_cuda, size_double, hipMemcpyDeviceToHost);
    hipMemcpy(cov2, cov2_cuda, size_double, hipMemcpyDeviceToHost);


    hipFree(ref_cuda);
    hipFree(cur_cuda);
    hipFree(depth_cuda);
    hipFree(cov2_cuda);
}
